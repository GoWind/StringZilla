#include "hip/hip_runtime.h"
/**
 *  @file   bench_similarity.cpp
 *  @brief  Benchmarks string similarity computations.
 *          It accepts a file with a list of words, and benchmarks the levenshtein edit-distance computations,
 *          alignment scores, and fingerprinting techniques combined with the Hamming distance.
 *
 *  Benchmarks include:
 *  - Linear-complexity basic & bounded Hamming distance computations.
 *  - Quadratic-complexity basic & bounded Levenshtein edit-distance computations.
 *  - Quadratic-complexity Needleman-Wunsch alignment scores for bioinformatics.
 *
 *  For Dynamic Programming algorithms, the number of operations per second are reported as the worst-case time
 *  complexity of the Cells Updates Per Second @b (CUPS) metric, meaning O(N*M) for a pair of strings with N and M
 *  characters, respectively.
 *
 *  Instead of CLI arguments, for compatibility with @b StringWa.rs, the following environment variables are used:
 *  - `STRINGWARS_DATASET` : Path to the dataset file.
 *  - `STRINGWARS_TOKENS=words` : Tokenization model ("file", "lines", "words", or positive integer [1:200] for N-grams
 *  - `STRINGWARS_SEED=42` : Optional seed for shuffling reproducibility.
 *
 *  Unlike StringWa.rs, the following additional environment variables are supported:
 *  - `STRINGWARS_DURATION=10` : Time limit (in seconds) per benchmark.
 *  - `STRINGWARS_STRESS=1` : Test SIMD-accelerated functions against the serial baselines.
 *  - `STRINGWARS_STRESS_DIR=/.tmp` : Output directory for stress-testing failures logs.
 *  - `STRINGWARS_STRESS_LIMIT=1` : Controls the number of failures we're willing to tolerate.
 *  - `STRINGWARS_STRESS_DURATION=10` : Stress-testing time limit (in seconds) per benchmark.
 *  - `STRINGWARS_FILTER` : Regular Expression pattern to filter algorithm/backend names.
 *
 *  Here are a few build & run commands:
 *
 *  @code{.sh}
 *  cmake -D STRINGZILLA_BUILD_BENCHMARK=1 -D CMAKE_BUILD_TYPE=Release -B build_release
 *  cmake --build build_release --config Release --target stringzilla_bench_similarity
 *  STRINGWARS_DATASET=xlsum.csv STRINGWARS_TOKENS=words build_release/stringzilla_bench_similarity
 *  @endcode
 *
 *  Alternatively, if you really want to stress-test a very specific function on a certain size inputs,
 *  like all Skylake-X and newer kernels on a boundary-condition input length of 64 bytes (exactly 1 cache line),
 *  your last command may look like:
 *
 *  @code{.sh}
 *  STRINGWARS_DATASET=proteins.txt STRINGWARS_TOKENS=64 STRINGWARS_FILTER=skylake
 *  STRINGWARS_STRESS=1 STRINGWARS_STRESS_DURATION=120 STRINGWARS_STRESS_DIR=logs
 *  build_release/stringzilla_bench_similarity
 *  @endcode
 *
 *  Unlike the full-blown StringWa.rs, it doesn't use any external frameworks like Criterion or Google Benchmark.
 *  This file is the sibling of `bench_search.cpp`, `bench_token.cpp`, `bench_sequence.cpp`, and `bench_memory.cpp`.
 */

#include "bench.hpp"
#include "test_stringcuzilla.cuh" // `levenshtein_baseline`, `error_costs_256x256_unary`

#if SZ_USE_CUDA
#include <stringcuzilla/similarity.cuh> // Parallel string processing on CUDA or OpenMP
#endif

#if SZ_USE_OPENMP
#include <stringcuzilla/similarity.hpp> // OpenMP templates for string similarity measures
#endif

namespace sz = ashvardanian::stringzilla;
using namespace sz::scripts;
using namespace std::literals; // for ""sv

using similarities_t = unified_vector<sz_ssize_t>;
using levenshtein_serial_t = sz::levenshtein_distances<sz_cap_parallel_k, char, std::allocator<char>>;
using levenshtein_cuda_t = sz::levenshtein_distances<sz_cap_cuda_k, char>;

#pragma region Levenshtein Distance and Alignment Scores

/** @brief Wraps a hardware-specific Levenshtein-distance backend into something @b `bench_unary`-compatible . */
template <typename engine_type_>
struct batch_callable {
    using engine_t = engine_type_;

    environment_t const &env;
    similarities_t &results;
    sz_size_t bound = SZ_SIZE_MAX;
    engine_t engine = {};

    batch_callable(environment_t const &env, similarities_t &res, sz_size_t batch_size) : env(env), results(res) {
        if (env.tokens.size() <= batch_size) throw std::runtime_error("Batch size is too large.");
    }

    inline call_result_t operator()(std::size_t batch_index) noexcept(false) {
        std::size_t const batch_size = results.size();
        std::size_t const forward_token_index = (batch_index * batch_size) % (env.tokens.size() - batch_size);
        std::size_t const backward_token_index = env.tokens.size() - forward_token_index - batch_size;

        return operator()({env.tokens.data() + forward_token_index, batch_size},
                          {env.tokens.data() + backward_token_index, batch_size});
    }

    inline call_result_t operator()(std::span<token_view_t const> a, std::span<token_view_t const> b) noexcept(false) {
        sz::status_t status = engine(a, b, results.data());
        if (status != sz::status_t::success_k) throw std::runtime_error("Failed to compute Levenshtein distance.");
        do_not_optimize(results);
        std::size_t bytes_passed = 0, cells_passed = 0;
        for (std::size_t i = 0; i < results.size(); ++i) {
            bytes_passed += std::min(a[i].size(), b[i].size());
            cells_passed += a[i].size() * b[i].size();
        }
        call_result_t call_result;
        call_result.bytes_passed = bytes_passed;
        call_result.operations = cells_passed;
        call_result.inputs_processed = results.size();
        call_result.check_value = reinterpret_cast<check_value_t>(&results);
        return call_result;
    }
};

struct similarities_equality_t {
    bool operator()(check_value_t const &a, check_value_t const &b) const {
        similarities_t const &a_ = *reinterpret_cast<similarities_t const *>(a);
        similarities_t const &b_ = *reinterpret_cast<similarities_t const *>(b);
        if (a_.size() != b_.size()) return false;
        for (std::size_t i = 0; i < a_.size(); ++i)
            if (a_[i] != b_[i]) {
                std::printf("Mismatch at index %zu: %zd != %zd\n", i, a_[i], b_[i]);
                return false;
            }
        return true;
    }
};

void bench_levenshtein(environment_t const &env) {

    std::vector<std::size_t> batch_sizes = {1024 / 32, 1024, 1024 * 32};
    similarities_t results_baseline, results_accelerated;

    for (std::size_t batch_size : batch_sizes) {
        results_baseline.resize(batch_size);
        results_accelerated.resize(batch_size);

        auto call_baseline = batch_callable<levenshtein_serial_t>(env, results_baseline, batch_size);
        auto name_baseline = "levenshtein_serial:batch"s + std::to_string(batch_size);
        bench_result_t baseline = bench_unary(env, name_baseline, call_baseline).log();

        bench_result_t accelerated =
            bench_unary(env, "levenshtein_cuda:batch"s + std::to_string(batch_size), call_baseline,
                        batch_callable<levenshtein_cuda_t>(env, results_accelerated, batch_size),
                        callable_no_op_t {},        // preprocessing
                        similarities_equality_t {}) // equality check
                .log(baseline);
    }
}

#pragma endregion

int main(int argc, char const **argv) {
    std::printf("Welcome to StringZilla!\n");

    try {
        std::printf("Building up the environment...\n");
        environment_t env = build_environment( //
            argc, argv,                        //
            "xlsum.csv",                       // Preferred for UTF-8 content
            environment_t::tokenization_t::lines_k);

        std::printf("Starting string similarity benchmarks...\n");
        bench_levenshtein(env);
    }
    catch (std::exception const &e) {
        std::fprintf(stderr, "Failed with: %s\n", e.what());
        return 1;
    }

    std::printf("All benchmarks finished.\n");
    return 0;
}