#include "hip/hip_runtime.h"
/**
 *  @brief   Extensive @b stress-testing suite for StringCuZilla parallel operations, written in CUDA C++.
 *  @see     Stress-tests on real-world and synthetic data are integrated into the @b `scripts/bench*.cpp` benchmarks.
 *
 *  @file    test.cu
 *  @author  Ash Vardanian
 */
#undef NDEBUG // ! Enable all assertions for testing

/**
 *  ! Overload the following with caution.
 *  ! Those parameters must never be explicitly set during releases,
 *  ! but they come handy during development, if you want to validate
 *  ! different ISA-specific implementations.

#define SZ_USE_HASWELL 0
#define SZ_USE_SKYLAKE 0
#define SZ_USE_ICE 0
#define SZ_USE_NEON 0
#define SZ_USE_SVE 0
 */
#if defined(SZ_DEBUG)
#undef SZ_DEBUG
#endif
#define SZ_DEBUG 1 // Enforce aggressive logging for this unit.

/**
 *  ! Overload the following with caution to enable parallelism.
 *  ! They control the OpenMP CPU backend as well as the CUDA GPU backend.
 */
#include <stringcuzilla/similarity.hpp>

#if SZ_USE_CUDA
#include <stringcuzilla/similarity.cuh>
#endif

#if !_SZ_IS_CPP17
#error "This test requires C++17 or later."
#endif

#include "test.hpp" // `levenshtein_baseline`

namespace sz = ashvardanian::stringzilla;
using namespace sz::scripts;
using namespace std::literals; // for ""sv

using arrow_strings_view_t = sz::arrow_strings_view<char, sz_size_t>;

#if !SZ_USE_CUDA
using arrow_strings_tape_t = sz::arrow_strings_tape<char, sz_size_t, std::allocator<char>>;
template <typename value_type_>
using unified_vector = std::vector<value_type_, std::allocator<value_type_>>;
#else
using arrow_strings_tape_t = sz::arrow_strings_tape<char, sz_size_t, sz::unified_alloc<char>>;
template <typename value_type_>
using unified_vector = std::vector<value_type_, sz::unified_alloc<value_type_>>;
#endif

struct levenshtein_baselines_t {
    template <typename results_type_>
    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, results_type_ *results) const {
        _sz_assert(first.size() == second.size());
#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::levenshtein_baseline(first[i].data(), first[i].size(), //
                                                           second[i].data(), second[i].size());
        return sz::status_t::success_k;
    }
};

struct needleman_wunsch_baselines_t {

    sz::error_costs_256x256_t substitution_costs = sz::error_costs_256x256_t::diagonal();
    sz::error_cost_t gap_cost = -1;

    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, sz_ssize_t *results) const {
        _sz_assert(first.size() == second.size());

#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::needleman_wunsch_baseline(first[i].data(), first[i].size(),   //
                                                                second[i].data(), second[i].size(), //
                                                                substitution_costs, gap_cost);
        return sz::status_t::success_k;
    }
};

struct smith_waterman_baselines_t {

    sz::error_costs_256x256_t substitution_costs = sz::error_costs_256x256_t::diagonal();
    sz::error_cost_t gap_cost = -1;

    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, sz_ssize_t *results) const {
        _sz_assert(first.size() == second.size());

#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::smith_waterman_baseline(first[i].data(), first[i].size(),   //
                                                              second[i].data(), second[i].size(), //
                                                              substitution_costs, gap_cost);
        return sz::status_t::success_k;
    }
};

using levenshtein_serial_t = sz::levenshtein_distances<sz_cap_parallel_k, char, std::allocator<char>>;
using levenshtein_utf8_serial_t = sz::levenshtein_distances_utf8<sz_cap_parallel_k, char, std::allocator<char>>;
using needleman_wunsch_serial_t = sz::needleman_wunsch_scores<sz_cap_parallel_k, char, std::allocator<char>>;
using smith_waterman_serial_t = sz::smith_waterman_scores<sz_cap_parallel_k, char, std::allocator<char>>;

/**
 *  In @b AVX-512:
 *  - for Global Alignments, we can vectorize the min-max calculation for diagonal "walkers"
 *  - for Local Alignments, we can vectorize the character substitution lookups for horizontal "walkers"
 */
using levenshtein_ice_t = sz::levenshtein_distances<sz_cap_ice_k, char, std::allocator<char>>;
using levenshtein_utf8_ice_t = sz::levenshtein_distances_utf8<sz_cap_ice_k, char, std::allocator<char>>;
using needleman_wunsch_ice_t = sz::needleman_wunsch_scores<sz_cap_ice_k, char, std::allocator<char>>;
using smith_waterman_ice_t = sz::smith_waterman_scores<sz_cap_ice_k, char, std::allocator<char>>;

/**
 *  In @b CUDA:
 *  - for GPUs before Hopper, we can use the @b SIMT model for warp-level parallelism using diagonal "walkers"
 *  - for GPUs after Hopper, we compound that with thread-level @b SIMD via @b DPX instructions for min-max
 */
using levenshtein_cuda_t = sz::levenshtein_distances<sz_cap_cuda_k, char>;
using levenshtein_utf8_cuda_t = sz::levenshtein_distances_utf8<sz_cap_cuda_k, char>;
using needleman_wunsch_cuda_t = sz::needleman_wunsch_scores<sz_cap_cuda_k, char>;
using smith_waterman_cuda_t = sz::smith_waterman_scores<sz_cap_cuda_k, char>;

using levenshtein_hopper_t = sz::levenshtein_distances<sz_cap_hopper_k, char>;
using levenshtein_utf8_hopper_t = sz::levenshtein_distances_utf8<sz_cap_hopper_k, char>;
using needleman_wunsch_hopper_t = sz::needleman_wunsch_scores<sz_cap_hopper_k, char>;
using smith_waterman_hopper_t = sz::smith_waterman_scores<sz_cap_hopper_k, char>;

template <typename score_type_>
void edit_distance_log_mismatch(std::string const &first, std::string const &second, //
                                score_type_ result_base, score_type_ result_simd) {
    char const *ellipsis = first.length() > 22 || second.length() > 22 ? "..." : "";
    char const *format_string;
    constexpr bool is_signed = std::is_signed<score_type_>();
    if constexpr (is_signed) {
        format_string = "Edit Distance error (got %zd, expected %zd): \"%.22s%s\" ⇔ \"%.22s%s\" \n";
    }
    else { format_string = "Edit Distance error (got %zu, expected %zu): \"%.22s%s\" ⇔ \"%.22s%s\" \n"; }
    std::printf(format_string, result_simd, result_base, first.c_str(), ellipsis, second.c_str(), ellipsis);
}

/**
 *  @brief  Tests the correctness of the string class Levenshtein distance computation,
 *          as well as the similarity scoring functions for bioinformatics-like workloads.
 */
template <typename score_type_, typename base_operator_, typename simd_operator_>
static void edit_distances_compare(base_operator_ &&base_operator, simd_operator_ &&simd_operator,
                                   std::size_t batch_size = 1024 * 16, std::size_t max_string_length = 512) {

    using score_t = score_type_;

    std::vector<std::pair<std::string, std::string>> test_cases = {
        {"ABC", "ABC"},                  // same string; distance ~ 0
        {"listen", "silent"},            // distance ~ 4
        {"atca", "ctactcaccc"},          // distance ~ 6
        {"A", "="},                      // distance ~ 1
        {"a", "a"},                      // distance ~ 0
        {"", ""},                        // distance ~ 0
        {"", "abc"},                     // distance ~ 3
        {"abc", ""},                     // distance ~ 3
        {"abc", "ac"},                   // one deletion; distance ~ 1
        {"abc", "a_bc"},                 // one insertion; distance ~ 1
        {"ggbuzgjux{}l", "gbuzgjux{}l"}, // one (prepended) insertion; distance ~ 1
        {"abc", "adc"},                  // one substitution; distance ~ 1
        {"apple", "aple"},               // distance ~ 1
        //
        // Unicode:
        {"αβγδ", "αγδ"},                      // Each Greek symbol is 2 bytes in size; 2 bytes, 1 runes diff.
        {"مرحبا بالعالم", "مرحبا يا عالم"},   // "Hello World" vs "Welcome to the World" ?; 3 bytes, 2 runes diff.
        {"école", "école"},                   // letter "é" as a single character vs "e" + "´"; 3 bytes, 2 runes diff.
        {"Schön", "Scho\u0308n"},             // "ö" represented as "o" + "¨"; 3 bytes, 2 runes diff.
        {"💖", "💗"},                         // 4-byte emojis: Different hearts; 1 bytes, 1 runes diff.
        {"𠜎 𠜱 𠝹 𠱓", "𠜎𠜱𠝹𠱓"},          // Ancient Chinese characters, no spaces vs spaces; 3 bytes, 3 runes
        {"München", "Muenchen"},              // German name with umlaut vs. its transcription; 2 bytes, 2 runes
        {"façade", "facade"},                 // "ç" represented as "c" with cedilla vs. plain "c"; 2 bytes, 1 runes
        {"こんにちは世界", "こんばんは世界"}, // "Good morning world" vs "Good evening world"; 3 bytes, 2 runes
        {"👩‍👩‍👧‍👦", "👨‍👩‍👧‍👦"}, // Different family emojis; 1 bytes, 1 runes
        {"Data科学123", "Data科學321"},                             // 3 bytes, 3 runes
        {"🙂🌍🚀", "🙂🌎✨"},                                       // 5 bytes, 2 runes
    };

    // First check with a batch-size of 1
    unified_vector<score_t> results_base(1), results_simd(1);
    arrow_strings_tape_t first_tape, second_tape;
    for (auto [first, second] : test_cases) {

        // Reset the tapes and results
        results_base[0] = 0, results_simd[0] = 0;
        first_tape.try_assign(&first, &first + 1);
        second_tape.try_assign(&second, &second + 1);

        // Compute with both backends
        sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
        sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
        _sz_assert(status_base == sz::status_t::success_k);
        _sz_assert(status_simd == sz::status_t::success_k);
        if (results_base[0] != results_simd[0])
            edit_distance_log_mismatch(first, second, results_base[0], results_simd[0]);
    }

    // Unzip the test cases into two separate tapes and perform batch processing
    results_base.resize(test_cases.size());
    results_simd.resize(test_cases.size());
    first_tape.reset();
    second_tape.reset();
    for (auto [first, second] : test_cases) {
        _sz_assert(first_tape.try_append({first.data(), first.size()}) == sz::status_t::success_k);
        _sz_assert(second_tape.try_append({second.data(), second.size()}) == sz::status_t::success_k);
    }

    // Compute with both backends
    sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
    sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
    _sz_assert(status_base == sz::status_t::success_k);
    _sz_assert(status_simd == sz::status_t::success_k);

    // Individually log the failed results
    for (std::size_t i = 0; i != test_cases.size(); ++i) {
        if (results_base[i] == results_simd[i]) continue;
        edit_distance_log_mismatch(test_cases[i].first, test_cases[i].second, results_base[i], results_simd[i]);
    }

    // Generate some random strings, using a small alphabet
    for (std::size_t iteration_idx = 0; iteration_idx < 10; ++iteration_idx) {
        std::vector<std::string> first_array(batch_size), second_array(batch_size);
        for (std::size_t i = 0; i != batch_size; ++i) {
            std::size_t first_length = 1u + std::rand() % max_string_length;
            std::size_t second_length = 1u + std::rand() % max_string_length;
            first_array[i] = random_string(first_length, "abc", 3);
            second_array[i] = random_string(second_length, "abc", 3);
        }

        // Convert to a GPU-friendly layout
        first_tape.try_assign(first_array.data(), first_array.data() + batch_size);
        second_tape.try_assign(second_array.data(), second_array.data() + batch_size);
        results_base.resize(batch_size);
        results_simd.resize(batch_size);

        // Compute with both backends
        sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
        sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
        _sz_assert(status_base == sz::status_t::success_k);
        _sz_assert(status_simd == sz::status_t::success_k);

        // Individually log the failed results
        for (std::size_t i = 0; i != test_cases.size(); ++i) {
            if (results_base[i] == results_simd[i]) continue;
            edit_distance_log_mismatch(first_array[i], second_array[i], results_base[i], results_simd[i]);
        }
    }
}

static void test_equivalence(std::size_t batch_size = 1024, std::size_t max_string_length = 100) {

    using error_t = sz::error_cost_t;
    using error_matrix_t = sz::error_costs_256x256_t; // ? Full matrix for all 256 ASCII characters
    using error_mat_t = sz::error_costs_26x26ascii_t; // ? Smaller compact form for 26 capital ASCII characters

    // Our logic of computing NW and SW alignment similarity scores differs in sign from most implementations.
    // It's similar to how the "cosine distance" is the inverse of the "cosine similarity".
    // In our case we compute the "distance" and by negating the sign, we can compute the "similarity".
    constexpr error_t unary_match_score = 1;
    constexpr error_t unary_mismatch_score = 0;
    constexpr error_t unary_gap_score = 0;
    error_matrix_t substituter_unary = error_matrix_t::diagonal(unary_match_score, unary_mismatch_score);
    {
        auto distance_l = levenshtein_baseline("abcdefg", 7, "abc_efg", 7);
        auto similarity_nw = needleman_wunsch_baseline("abcdefg", 7, "abc_efg", 7, substituter_unary, unary_gap_score);
        auto similarity_sw = smith_waterman_baseline("abcdefg", 7, "abc_efg", 7, substituter_unary, unary_gap_score);
        // Distance can be computed from the similarity, by inverting the sign around the length of the longest string:
        auto distance_nw = std::max(7, 7) - similarity_nw;
        auto distance_sw = std::max(7, 7) - similarity_sw;
        _sz_assert(distance_l == 1);
        _sz_assert(distance_nw == 1);
        _sz_assert(distance_sw == 1);
    }

    // Now systematically compare the results of the baseline and SIMD implementations
    constexpr sz_capability_t serial_k = sz_cap_serial_k;
    constexpr sz_capability_t parallel_k = sz_cap_parallel_k;
    constexpr sz_capability_t cuda_k = sz_cap_cuda_k;

    // Single-threaded serial Levenshtein distance implementation
    edit_distances_compare<sz_size_t>(                                      //
        levenshtein_baselines_t {},                                         //
        sz::levenshtein_distances<serial_k, char, std::allocator<char>> {}, //
        batch_size, max_string_length);

    // Multi-threaded parallel Levenshtein distance implementation
    edit_distances_compare<sz_size_t>(                                        //
        levenshtein_baselines_t {},                                           //
        sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {}, //
        batch_size, max_string_length);

    // Now let's take non-unary substitution costs, like BLOSUM62
    constexpr error_t blosum62_gap_extension_cost = 4; // ? The inverted typical (-4) value
    error_matrix_t blosum62 = sz::error_costs_26x26ascii_t::blosum62().decompressed();

    // Single-threaded serial NW implementation
    edit_distances_compare<sz_ssize_t>(                                       //
        needleman_wunsch_baselines_t {blosum62, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<serial_k, char, error_matrix_t, std::allocator<char>> {
            blosum62, blosum62_gap_extension_cost}, //
        batch_size, max_string_length);

    // Multi-threaded parallel NW implementation
    edit_distances_compare<sz_ssize_t>(                                       //
        needleman_wunsch_baselines_t {blosum62, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62, blosum62_gap_extension_cost}, //
        batch_size, max_string_length);

    // Single-threaded serial SW implementation
    edit_distances_compare<sz_ssize_t>(                                     //
        smith_waterman_baselines_t {blosum62, blosum62_gap_extension_cost}, //
        sz::smith_waterman_scores<serial_k, char, error_matrix_t, std::allocator<char>> {
            blosum62, blosum62_gap_extension_cost}, //
        batch_size, max_string_length);

    // Multi-threaded parallel SW implementation
    edit_distances_compare<sz_ssize_t>(                                     //
        smith_waterman_baselines_t {blosum62, blosum62_gap_extension_cost}, //
        sz::smith_waterman_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62, blosum62_gap_extension_cost}, //
        batch_size, max_string_length);

    // Switch to the GPU, using an identical matrix, but move it into unified memory
    unified_vector<error_matrix_t> blosum62_unified(1);
    blosum62_unified[0] = blosum62;

    // CUDA Levenshtein distance against Multi-threaded on CPU
    edit_distances_compare<sz_size_t>(                                        //
        sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {}, //
        sz::levenshtein_distances<cuda_k, char> {},                           //
        batch_size, max_string_length);

    // CUDA Needleman-Wunsch distance against Multi-threaded on CPU
    edit_distances_compare<sz_ssize_t>( //
        sz::needleman_wunsch_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<cuda_k, char, error_matrix_t *> {blosum62_unified.data(),
                                                                     blosum62_gap_extension_cost},
        batch_size, max_string_length);
};

#if 0
/**
 *  @brief  Invokes different C++ member methods of immutable strings to cover
 *          extensions beyond the STL API.
 */
template <typename string_type>
static void test_non_stl_extensions_for_reads() {
    using str = string_type;

    // Computing edit-distances.
    _sz_assert(sz::hamming_distance(str("hello"), str("hello")) == 0);
    _sz_assert(sz::hamming_distance(str("hello"), str("hell")) == 1);
    _sz_assert(sz::hamming_distance(str("abc"), str("adc")) == 1);                // one substitution
    _sz_assert(sz::hamming_distance(str("αβγδ"), str("αxxγδ")) == 2);             // replace Beta UTF8 codepoint
    _sz_assert(sz::hamming_distance_utf8(str("abcdefgh"), str("_bcdefg_")) == 2); // replace ASCI prefix and suffix
    _sz_assert(sz::hamming_distance_utf8(str("αβγδ"), str("αγγδ")) == 1);         // replace Beta UTF8 codepoint

    _sz_assert(sz::levenshtein_distance(str("hello"), str("hello")) == 0);
    _sz_assert(sz::levenshtein_distance(str("hello"), str("hell")) == 1);
    _sz_assert(sz::levenshtein_distance(str(""), str("")) == 0);
    _sz_assert(sz::levenshtein_distance(str(""), str("abc")) == 3);
    _sz_assert(sz::levenshtein_distance(str("abc"), str("")) == 3);
    _sz_assert(sz::levenshtein_distance(str("abc"), str("ac")) == 1);                   // one deletion
    _sz_assert(sz::levenshtein_distance(str("abc"), str("a_bc")) == 1);                 // one insertion
    _sz_assert(sz::levenshtein_distance(str("abc"), str("adc")) == 1);                  // one substitution
    _sz_assert(sz::levenshtein_distance(str("ggbuzgjux{}l"), str("gbuzgjux{}l")) == 1); // one insertion (prepended)
    _sz_assert(sz::levenshtein_distance(str("abcdefgABCDEFG"), str("ABCDEFGabcdefg")) == 14);

    _sz_assert(sz::levenshtein_distance_utf8(str("hello"), str("hell")) == 1);           // no unicode symbols, just ASCII
    _sz_assert(sz::levenshtein_distance_utf8(str("𠜎 𠜱 𠝹 𠱓"), str("𠜎𠜱𠝹𠱓")) == 3); // add 3 whitespaces in Chinese
    _sz_assert(sz::levenshtein_distance_utf8(str("💖"), str("💗")) == 1);

    _sz_assert(sz::levenshtein_distance_utf8(str("αβγδ"), str("αγδ")) == 1); // insert Beta
    _sz_assert(sz::levenshtein_distance_utf8(str("école"), str("école")) ==
           2); // etter "é" as a single character vs "e" + "´"
    _sz_assert(sz::levenshtein_distance_utf8(str("façade"), str("facade")) == 1);     // "ç" with cedilla vs. plain
    _sz_assert(sz::levenshtein_distance_utf8(str("Schön"), str("Scho\u0308n")) == 2); // "ö" represented as "o" + "¨"
    _sz_assert(sz::levenshtein_distance_utf8(str("München"), str("Muenchen")) == 2); // German with umlaut vs. transcription
    _sz_assert(sz::levenshtein_distance_utf8(str("こんにちは世界"), str("こんばんは世界")) == 2);

    // Computing alignment scores.
    using matrix_t = std::int8_t[256][256];
    sz::error_costs_256x256_t substitution_costs = error_costs_256x256_diagonal();
    matrix_t &costs = *reinterpret_cast<matrix_t *>(substitution_costs.data());

    _sz_assert(sz::alignment_score(str("listen"), str("silent"), costs, -1) == -4);
    _sz_assert(sz::alignment_score(str("abcdefgABCDEFG"), str("ABCDEFGabcdefg"), costs, -1) == -14);
    _sz_assert(sz::alignment_score(str("hello"), str("hello"), costs, -1) == 0);
    _sz_assert(sz::alignment_score(str("hello"), str("hell"), costs, -1) == -1);

    // Computing rolling fingerprints.
    _sz_assert(sz::hashes_fingerprint<512>(str("aaaa"), 3).count() == 1);
    _sz_assert(sz::hashes_fingerprint<512>(str("hello"), 4).count() == 2);
    _sz_assert(sz::hashes_fingerprint<512>(str("hello"), 3).count() == 3);

    // No matter how many times one repeats a character, the hash should only contain at most one set bit.
    _sz_assert(sz::hashes_fingerprint<512>(str("a"), 3).count() == 0);
    _sz_assert(sz::hashes_fingerprint<512>(str("aa"), 3).count() == 0);
    _sz_assert(sz::hashes_fingerprint<512>(str("aaa"), 3).count() == 1);
    _sz_assert(sz::hashes_fingerprint<512>(str("aaaa"), 3).count() == 1);
    _sz_assert(sz::hashes_fingerprint<512>(str("aaaaa"), 3).count() == 1);

    // Computing fuzzy search results.
}
#endif

int main(int argc, char const **argv) {

    // Let's greet the user nicely
    sz_unused(argc && argv);
    std::printf("Hi, dear tester! You look nice today!\n");
    std::printf("- Uses Haswell: %s \n", SZ_USE_HASWELL ? "yes" : "no");
    std::printf("- Uses Skylake: %s \n", SZ_USE_SKYLAKE ? "yes" : "no");
    std::printf("- Uses Ice Lake: %s \n", SZ_USE_ICE ? "yes" : "no");
    std::printf("- Uses NEON: %s \n", SZ_USE_NEON ? "yes" : "no");
    std::printf("- Uses SVE: %s \n", SZ_USE_SVE ? "yes" : "no");
    std::printf("- Uses SVE2: %s \n", SZ_USE_SVE2 ? "yes" : "no");
    std::printf("- Uses OpenMP: %s \n", SZ_USE_OPENMP ? "yes" : "no");
    std::printf("- Uses CUDA: %s \n", SZ_USE_CUDA ? "yes" : "no");

#if SZ_USE_CUDA
    hipError_t cuda_error = hipFree(0); // Force context initialization
    if (cuda_error != hipSuccess) {
        std::printf("CUDA initialization error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    int device_count = 0;
    cuda_error = hipGetDeviceCount(&device_count);
    if (cuda_error != hipSuccess) {
        std::printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    std::printf("CUDA device count: %d\n", device_count);
    if (device_count == 0) {
        std::printf("No CUDA devices found.\n");
        return 1;
    }
    std::printf("- CUDA devices:\n");
    hipDeviceProp_t prop;
    for (int i = 0; i < device_count; ++i) {
        cuda_error = hipGetDeviceProperties(&prop, i);
        std::printf("  - %s\n", prop.name);
    }
    std::printf("- CUDA managed memory support: %s\n", prop.managedMemory == 1 ? "yes" : "no");
    std::printf("- CUDA unified memory support: %s\n", prop.unifiedAddressing == 1 ? "yes" : "no");
#endif

    test_equivalence();

    std::printf("All tests passed... Unbelievable!\n");
    return 0;
}
