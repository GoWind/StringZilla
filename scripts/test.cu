#include "hip/hip_runtime.h"
/**
 *  @brief   Extensive @b stress-testing suite for StringCuZilla parallel operations, written in CUDA C++.
 *  @see     Stress-tests on real-world and synthetic data are integrated into the @b `scripts/bench*.cpp` benchmarks.
 *
 *  @file    test.cu
 *  @author  Ash Vardanian
 */
#undef NDEBUG // ! Enable all assertions for testing

/**
 *  ! Overload the following with caution.
 *  ! Those parameters must never be explicitly set during releases,
 *  ! but they come handy during development, if you want to validate
 *  ! different ISA-specific implementations.

#define SZ_USE_HASWELL 0
#define SZ_USE_SKYLAKE 0
#define SZ_USE_ICE 0
#define SZ_USE_NEON 0
#define SZ_USE_SVE 0
 */
#if defined(SZ_DEBUG)
#undef SZ_DEBUG
#endif
#define SZ_DEBUG 1 // Enforce aggressive logging for this unit.

/**
 *  ! Overload the following with caution to enable parallelism.
 *  ! They control the OpenMP CPU backend as well as the CUDA GPU backend.
 */
#include "stringcuzilla/similarity.hpp"

#if SZ_USE_CUDA
#include "stringcuzilla/similarity.cuh"
#endif

#if !_SZ_IS_CPP17
#error "This test requires C++17 or later."
#endif

#include "test.hpp" // `levenshtein_baseline`

namespace sz = ashvardanian::stringzilla;
using namespace sz::scripts;
using namespace std::literals; // for ""sv

struct levenshtein_baselines_t {
    template <typename results_type_>
    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, results_type_ *results) const {
        _sz_assert(first.size() == second.size());
#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::levenshtein_baseline(first[i].data(), first[i].size(), //
                                                           second[i].data(), second[i].size());
        return sz::status_t::success_k;
    }
};

struct needleman_wunsch_baselines_t {

    sz::error_costs_256x256_t substitution_costs = sz::error_costs_256x256_t::diagonal();
    sz::error_cost_t gap_cost = -1;

    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, sz_ssize_t *results) const {
        _sz_assert(first.size() == second.size());

#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::needleman_wunsch_baseline(first[i].data(), first[i].size(),   //
                                                                second[i].data(), second[i].size(), //
                                                                substitution_costs, gap_cost);
        return sz::status_t::success_k;
    }
};

struct smith_waterman_baselines_t {

    sz::error_costs_256x256_t substitution_costs = sz::error_costs_256x256_t::diagonal();
    sz::error_cost_t gap_cost = -1;

    sz::status_t operator()(arrow_strings_view_t first, arrow_strings_view_t second, sz_ssize_t *results) const {
        _sz_assert(first.size() == second.size());

#pragma omp parallel for
        for (std::size_t i = 0; i != first.size(); ++i)
            results[i] = sz::scripts::smith_waterman_baseline(first[i].data(), first[i].size(),   //
                                                              second[i].data(), second[i].size(), //
                                                              substitution_costs, gap_cost);
        return sz::status_t::success_k;
    }
};

using levenshtein_serial_t = sz::levenshtein_distances<sz_cap_parallel_k, char, std::allocator<char>>;
using levenshtein_utf8_serial_t = sz::levenshtein_distances_utf8<sz_cap_parallel_k, char, std::allocator<char>>;
using needleman_wunsch_serial_t = sz::needleman_wunsch_scores<sz_cap_parallel_k, char, std::allocator<char>>;
using smith_waterman_serial_t = sz::smith_waterman_scores<sz_cap_parallel_k, char, std::allocator<char>>;

/**
 *  In @b AVX-512:
 *  - for Global Alignments, we can vectorize the min-max calculation for diagonal "walkers"
 *  - for Local Alignments, we can vectorize the character substitution lookups for horizontal "walkers"
 */
using levenshtein_ice_t = sz::levenshtein_distances<sz_cap_ice_k, char, std::allocator<char>>;
using levenshtein_utf8_ice_t = sz::levenshtein_distances_utf8<sz_cap_ice_k, char, std::allocator<char>>;
using needleman_wunsch_ice_t = sz::needleman_wunsch_scores<sz_cap_ice_k, char, std::allocator<char>>;
using smith_waterman_ice_t = sz::smith_waterman_scores<sz_cap_ice_k, char, std::allocator<char>>;

/**
 *  In @b CUDA:
 *  - for GPUs before Hopper, we can use the @b SIMT model for warp-level parallelism using diagonal "walkers"
 *  - for GPUs after Hopper, we compound that with thread-level @b SIMD via @b DPX instructions for min-max
 */
using levenshtein_cuda_t = sz::levenshtein_distances<sz_cap_cuda_k, char>;
using levenshtein_utf8_cuda_t = sz::levenshtein_distances_utf8<sz_cap_cuda_k, char>;
using needleman_wunsch_cuda_t = sz::needleman_wunsch_scores<sz_cap_cuda_k, char>;
using smith_waterman_cuda_t = sz::smith_waterman_scores<sz_cap_cuda_k, char>;

using levenshtein_hopper_t = sz::levenshtein_distances<sz_cap_hopper_k, char>;
using levenshtein_utf8_hopper_t = sz::levenshtein_distances_utf8<sz_cap_hopper_k, char>;
using needleman_wunsch_hopper_t = sz::needleman_wunsch_scores<sz_cap_hopper_k, char>;
using smith_waterman_hopper_t = sz::smith_waterman_scores<sz_cap_hopper_k, char>;

template <typename score_type_>
void edit_distance_log_mismatch(std::string const &first, std::string const &second, //
                                score_type_ result_base, score_type_ result_simd) {
    char const *ellipsis = first.length() > 22 || second.length() > 22 ? "..." : "";
    char const *format_string;
    constexpr bool is_signed = std::is_signed<score_type_>();
    if constexpr (is_signed) {
        format_string = "Edit Distance error (got %zd, expected %zd): \"%.22s%s\" ⇔ \"%.22s%s\" \n";
    }
    else { format_string = "Edit Distance error (got %zu, expected %zu): \"%.22s%s\" ⇔ \"%.22s%s\" \n"; }
    std::printf(format_string, result_simd, result_base, first.c_str(), ellipsis, second.c_str(), ellipsis);
}

/**
 *  @brief  Tests the correctness of the string class Levenshtein distance computation,
 *          as well as the similarity scoring functions for bioinformatics-like workloads
 *          on a @b fixed set of different representative ASCII and UTF-8 strings.
 */
template <typename score_type_, typename base_operator_, typename simd_operator_>
static void edit_distances_fixed(base_operator_ &&base_operator, simd_operator_ &&simd_operator,
                                 std::string_view allowed_chars = {}) {

    std::vector<std::pair<std::string, std::string>> test_cases = {
        {"ABC", "ABC"},                  // same string; distance ~ 0
        {"LISTEN", "SILENT"},            // distance ~ 4
        {"ATCA", "CTACTCACCC"},          // distance ~ 6
        {"A", "="},                      // distance ~ 1
        {"A", "A"},                      // distance ~ 0
        {"", ""},                        // distance ~ 0
        {"", "ABC"},                     // distance ~ 3
        {"ABC", ""},                     // distance ~ 3
        {"ABC", "AC"},                   // one deletion; distance ~ 1
        {"ABC", "A_BC"},                 // one insertion; distance ~ 1
        {"ggbuzgjux{}l", "gbuzgjux{}l"}, // one (prepended) insertion; distance ~ 1
        {"ABC", "ADC"},                  // one substitution; distance ~ 1
        {"APPLE", "APLE"},               // distance ~ 1
        //
        // Unicode:
        {"αβγδ", "αγδ"},                      // Each Greek symbol is 2 bytes in size; 2 bytes, 1 runes diff.
        {"مرحبا بالعالم", "مرحبا يا عالم"},   // "Hello World" vs "Welcome to the World" ?; 3 bytes, 2 runes diff.
        {"école", "école"},                   // letter "é" as a single character vs "e" + "´"; 3 bytes, 2 runes diff.
        {"Schön", "Scho\u0308n"},             // "ö" represented as "o" + "¨"; 3 bytes, 2 runes diff.
        {"💖", "💗"},                         // 4-byte emojis: Different hearts; 1 bytes, 1 runes diff.
        {"𠜎 𠜱 𠝹 𠱓", "𠜎𠜱𠝹𠱓"},          // Ancient Chinese characters, no spaces vs spaces; 3 bytes, 3 runes
        {"München", "Muenchen"},              // German name with umlaut vs. its transcription; 2 bytes, 2 runes
        {"façade", "facade"},                 // "ç" represented as "c" with cedilla vs. plain "c"; 2 bytes, 1 runes
        {"こんにちは世界", "こんばんは世界"}, // "Good morning world" vs "Good evening world"; 3 bytes, 2 runes
        {"👩‍👩‍👧‍👦", "👨‍👩‍👧‍👦"}, // Different family emojis; 1 bytes, 1 runes
        {"Data科学123", "Data科學321"},                             // 3 bytes, 3 runes
        {"🙂🌍🚀", "🙂🌎✨"},                                       // 5 bytes, 2 runes
    };

    // First check with a batch-size of 1
    using score_t = score_type_;
    unified_vector<score_t> results_base(1), results_simd(1);
    arrow_strings_tape_t first_tape, second_tape;
    bool contains_missing_in_any_case = false;
    for (auto [first, second] : test_cases) {

        // Check if the input strings fit into our allowed characters set
        if (!allowed_chars.empty()) {
            bool contains_missing = false;
            for (auto c : first) contains_missing |= allowed_chars.find(c) == std::string_view::npos;
            for (auto c : second) contains_missing |= allowed_chars.find(c) == std::string_view::npos;
            contains_missing_in_any_case |= contains_missing;
            if (contains_missing) continue;
        }

        // Reset the tapes and results
        results_base[0] = 0, results_simd[0] = 0;
        first_tape.try_assign(&first, &first + 1);
        second_tape.try_assign(&second, &second + 1);

        // Compute with both backends
        sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
        sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
        _sz_assert(status_base == sz::status_t::success_k);
        _sz_assert(status_simd == sz::status_t::success_k);
        if (results_base[0] != results_simd[0])
            edit_distance_log_mismatch(first, second, results_base[0], results_simd[0]);
    }

    // Unzip the test cases into two separate tapes and perform batch processing
    if (!contains_missing_in_any_case) {
        results_base.resize(test_cases.size());
        results_simd.resize(test_cases.size());
        first_tape.reset();
        second_tape.reset();
        for (auto [first, second] : test_cases) {
            _sz_assert(first_tape.try_append({first.data(), first.size()}) == sz::status_t::success_k);
            _sz_assert(second_tape.try_append({second.data(), second.size()}) == sz::status_t::success_k);
        }

        // Compute with both backends
        sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
        sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
        _sz_assert(status_base == sz::status_t::success_k);
        _sz_assert(status_simd == sz::status_t::success_k);

        // Individually log the failed results
        for (std::size_t i = 0; i != test_cases.size(); ++i) {
            if (results_base[i] == results_simd[i]) continue;
            edit_distance_log_mismatch(test_cases[i].first, test_cases[i].second, results_base[i], results_simd[i]);
        }
    }
}

struct fuzzy_config_t {
    std::string_view alphabet = "ABC";
    std::size_t batch_size = 1024 * 16;
    std::size_t min_string_length = 1;
    std::size_t max_string_length = 512;
    std::size_t iterations = 10;
};

/**
 *  @brief  Tests the correctness of the string class Levenshtein distance computation,
 *          as well as the similarity scoring functions for bioinformatics-like workloads
 *          on a synthetic @b randomly-generated set of strings from a given @p alphabet.
 */
template <typename score_type_, typename base_operator_, typename simd_operator_>
static void edit_distances_fuzzy(base_operator_ &&base_operator, simd_operator_ &&simd_operator,
                                 fuzzy_config_t config = {}) {

    using score_t = score_type_;
    unified_vector<score_t> results_base(config.batch_size), results_simd(config.batch_size);
    std::vector<std::string> first_array(config.batch_size), second_array(config.batch_size);
    arrow_strings_tape_t first_tape, second_tape;
    std::uniform_int_distribution<std::size_t> length_distribution(config.min_string_length, config.max_string_length);

    // Generate some random strings, using a small alphabet
    for (std::size_t iteration_idx = 0; iteration_idx < config.iterations; ++iteration_idx) {
        for (std::size_t i = 0; i != config.batch_size; ++i) {
            std::size_t first_length = length_distribution(global_random_generator());
            std::size_t second_length = length_distribution(global_random_generator());
            first_array[i] = random_string(first_length, config.alphabet.data(), config.alphabet.size());
            second_array[i] = random_string(second_length, config.alphabet.data(), config.alphabet.size());
        }

        // Convert to a GPU-friendly layout
        first_tape.try_assign(first_array.data(), first_array.data() + config.batch_size);
        second_tape.try_assign(second_array.data(), second_array.data() + config.batch_size);

        // Compute with both backends
        sz::status_t status_base = base_operator(first_tape.view(), second_tape.view(), results_base.data());
        sz::status_t status_simd = simd_operator(first_tape.view(), second_tape.view(), results_simd.data());
        _sz_assert(status_base == sz::status_t::success_k);
        _sz_assert(status_simd == sz::status_t::success_k);

        // Individually log the failed results
        for (std::size_t i = 0; i != config.batch_size; ++i) {
            if (results_base[i] == results_simd[i]) continue;
            edit_distance_log_mismatch(first_array[i], second_array[i], results_base[i], results_simd[i]);
        }
    }
}

template <typename score_type_, typename base_operator_, typename simd_operator_>
static void edit_distances_fixed_and_fuzzy(base_operator_ &&base_operator, simd_operator_ &&simd_operator,
                                           std::string_view allowed_chars = {}, fuzzy_config_t config = {}) {
    edit_distances_fixed<score_type_>(base_operator, simd_operator, allowed_chars);
    edit_distances_fuzzy<score_type_>(base_operator, simd_operator, config);
}

/**
 *  @brief  Tests the correctness of the string class Levenshtein distance, NW & SW score computation,
 *          comparing the results to some baseline implementation for predefined and random inputs.
 */
static void test_equivalence() {

    using error_t = sz::error_cost_t;
    using error_matrix_t = sz::error_costs_256x256_t; // ? Full matrix for all 256 ASCII characters
    using error_mat_t = sz::error_costs_26x26ascii_t; // ? Smaller compact form for 26 capital ASCII characters

    // Our logic of computing NW and SW alignment similarity scores differs in sign from most implementations.
    // It's similar to how the "cosine distance" is the inverse of the "cosine similarity".
    // In our case we compute the "distance" and by negating the sign, we can compute the "similarity".
    constexpr error_t unary_match_score = 1;
    constexpr error_t unary_mismatch_score = 0;
    constexpr error_t unary_gap_score = 0;
    error_matrix_t substituter_unary = error_matrix_t::diagonal(unary_match_score, unary_mismatch_score);
    {
        auto distance_l = levenshtein_baseline("abcdefg", 7, "abc_efg", 7);
        auto similarity_nw = needleman_wunsch_baseline("abcdefg", 7, "abc_efg", 7, substituter_unary, unary_gap_score);
        auto similarity_sw = smith_waterman_baseline("abcdefg", 7, "abc_efg", 7, substituter_unary, unary_gap_score);
        // Distance can be computed from the similarity, by inverting the sign around the length of the longest string:
        auto distance_nw = std::max(7, 7) - similarity_nw;
        auto distance_sw = std::max(7, 7) - similarity_sw;
        _sz_assert(distance_l == 1);
        _sz_assert(distance_nw == 1);
        _sz_assert(distance_sw == 1);
    }

    // Now systematically compare the results of the baseline and SIMD implementations
    constexpr sz_capability_t serial_k = sz_cap_serial_k;
    constexpr sz_capability_t parallel_k = sz_cap_parallel_k;
    constexpr sz_capability_t cuda_k = sz_cap_cuda_k;
    constexpr sz_capability_t hopper_k = sz_cap_hopper_k;

    // Single-threaded serial Levenshtein distance implementation
    edit_distances_fixed_and_fuzzy<sz_size_t>( //
        levenshtein_baselines_t {},            //
        sz::levenshtein_distances<serial_k, char, std::allocator<char>> {});

    // Multi-threaded parallel Levenshtein distance implementation
    edit_distances_fixed_and_fuzzy<sz_size_t>( //
        levenshtein_baselines_t {},            //
        sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {});

    // Now let's take non-unary substitution costs, like BLOSUM62
    constexpr error_t blosum62_gap_extension_cost = -4;
    error_mat_t blosum62_mat = sz::error_costs_26x26ascii_t::blosum62();
    error_matrix_t blosum62_matrix = blosum62_mat.decompressed();

    // Single-threaded serial NW implementation
    edit_distances_fixed_and_fuzzy<sz_ssize_t>(                                      //
        needleman_wunsch_baselines_t {blosum62_matrix, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<serial_k, char, error_matrix_t, std::allocator<char>> {
            blosum62_matrix, blosum62_gap_extension_cost});

    // Multi-threaded parallel NW implementation
    edit_distances_fixed_and_fuzzy<sz_ssize_t>(                                      //
        needleman_wunsch_baselines_t {blosum62_matrix, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62_matrix, blosum62_gap_extension_cost});

    // Single-threaded serial SW implementation
    edit_distances_fixed_and_fuzzy<sz_ssize_t>(                                    //
        smith_waterman_baselines_t {blosum62_matrix, blosum62_gap_extension_cost}, //
        sz::smith_waterman_scores<serial_k, char, error_matrix_t, std::allocator<char>> {blosum62_matrix,
                                                                                         blosum62_gap_extension_cost});

    // Multi-threaded parallel SW implementation
    edit_distances_fixed_and_fuzzy<sz_ssize_t>(                                    //
        smith_waterman_baselines_t {blosum62_matrix, blosum62_gap_extension_cost}, //
        sz::smith_waterman_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62_matrix, blosum62_gap_extension_cost});

    // Switch to the GPU, using an identical matrix, but move it into unified memory
    unified_vector<error_mat_t> blosum62_unified(1);
    blosum62_unified[0] = blosum62_mat;

    // CUDA Levenshtein distance against Multi-threaded on CPU
    edit_distances_fixed_and_fuzzy<sz_size_t>(                                //
        sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {}, //
        sz::levenshtein_distances<cuda_k, char> {});

#if SZ_USE_HOPPER
    // CUDA Levenshtein distance on Hopper against Multi-threaded on CPU
    edit_distances_fixed_and_fuzzy<sz_size_t>(                                //
        sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {}, //
        sz::levenshtein_distances<hopper_k, char> {});
#endif

    // CUDA Needleman-Wunsch distance against Multi-threaded on CPU,
    // using a compressed smaller matrix to fit into GPU shared memory
    std::string_view ascii_alphabet = "ABCDEFGHIJKLMNOPQRSTUVWXYZ";
    edit_distances_fixed_and_fuzzy<sz_ssize_t>( //
        sz::needleman_wunsch_scores<parallel_k, char, error_matrix_t, std::allocator<char>> {
            blosum62_matrix, blosum62_gap_extension_cost}, //
        sz::needleman_wunsch_scores<cuda_k, char, error_mat_t *> {blosum62_unified.data(), blosum62_gap_extension_cost},
        ascii_alphabet);
}

/**
 *  @brief  Many GPU algorithms depend on effective use of shared memory and scheduling its allocation for
 *          long inputs or very large batches isn't trivial.
 */
void test_growing_memory_usage() {

    // Now systematically compare the results of the baseline and SIMD implementations
    constexpr sz_capability_t serial_k = sz_cap_serial_k;
    constexpr sz_capability_t parallel_k = sz_cap_parallel_k;
    constexpr sz_capability_t cuda_k = sz_cap_cuda_k;
    constexpr sz_capability_t hopper_k = sz_cap_hopper_k;

    std::vector<fuzzy_config_t> experiments = {
        // Single string pair of same length:
        {.batch_size = 1, .min_string_length = 512, .max_string_length = 512, .iterations = 1},
        {.batch_size = 1, .min_string_length = 2048, .max_string_length = 2048, .iterations = 1},
        {.batch_size = 1, .min_string_length = 8192, .max_string_length = 8192, .iterations = 1},
        {.batch_size = 1, .min_string_length = 32768, .max_string_length = 32768, .iterations = 1},
        {.batch_size = 1, .min_string_length = 131072, .max_string_length = 131072, .iterations = 1},
        // Two strings of a same length:
        {.batch_size = 2, .min_string_length = 512, .max_string_length = 512, .iterations = 1},
        {.batch_size = 2, .min_string_length = 2048, .max_string_length = 2048, .iterations = 1},
        {.batch_size = 2, .min_string_length = 8192, .max_string_length = 8192, .iterations = 1},
        {.batch_size = 2, .min_string_length = 32768, .max_string_length = 32768, .iterations = 1},
        {.batch_size = 2, .min_string_length = 131072, .max_string_length = 131072, .iterations = 1},
        // Ten strings of random lengths:
        {.batch_size = 10, .min_string_length = 1, .max_string_length = 512, .iterations = 1},
        {.batch_size = 10, .min_string_length = 1, .max_string_length = 2048, .iterations = 1},
        {.batch_size = 10, .min_string_length = 1, .max_string_length = 8192, .iterations = 1},
        {.batch_size = 10, .min_string_length = 1, .max_string_length = 32768, .iterations = 1},
        {.batch_size = 10, .min_string_length = 1, .max_string_length = 131072, .iterations = 1},
    };

    // Progress until something fails
    for (fuzzy_config_t const &experiment : experiments) {
        std::printf("Testing with batch size %zu, min length %zu, max length %zu, iterations %zu\n",
                    experiment.batch_size, experiment.min_string_length, experiment.max_string_length,
                    experiment.iterations);

        // Single-threaded serial Levenshtein distance implementation
        edit_distances_fuzzy<sz_size_t>( //
            levenshtein_baselines_t {},  //
            sz::levenshtein_distances<serial_k, char, std::allocator<char>> {}, experiment);

        // Multi-threaded parallel Levenshtein distance implementation
        edit_distances_fuzzy<sz_size_t>( //
            levenshtein_baselines_t {},  //
            sz::levenshtein_distances<parallel_k, char, std::allocator<char>> {}, experiment);
    }
}

int main(int argc, char const **argv) {

    // Let's greet the user nicely
    sz_unused(argc && argv);
    std::printf("Hi, dear tester! You look nice today!\n");
    std::printf("- Uses Haswell: %s \n", SZ_USE_HASWELL ? "yes" : "no");
    std::printf("- Uses Skylake: %s \n", SZ_USE_SKYLAKE ? "yes" : "no");
    std::printf("- Uses Ice Lake: %s \n", SZ_USE_ICE ? "yes" : "no");
    std::printf("- Uses NEON: %s \n", SZ_USE_NEON ? "yes" : "no");
    std::printf("- Uses SVE: %s \n", SZ_USE_SVE ? "yes" : "no");
    std::printf("- Uses SVE2: %s \n", SZ_USE_SVE2 ? "yes" : "no");
    std::printf("- Uses OpenMP: %s \n", SZ_USE_OPENMP ? "yes" : "no");
    std::printf("- Uses CUDA: %s \n", SZ_USE_CUDA ? "yes" : "no");

#if SZ_USE_CUDA
    hipError_t cuda_error = hipFree(0); // Force context initialization
    if (cuda_error != hipSuccess) {
        std::printf("CUDA initialization error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    int device_count = 0;
    cuda_error = hipGetDeviceCount(&device_count);
    if (cuda_error != hipSuccess) {
        std::printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    std::printf("CUDA device count: %d\n", device_count);
    if (device_count == 0) {
        std::printf("No CUDA devices found.\n");
        return 1;
    }
    std::printf("- CUDA devices:\n");
    hipDeviceProp_t prop;
    for (int i = 0; i < device_count; ++i) {
        cuda_error = hipGetDeviceProperties(&prop, i);
        std::printf("  - %s\n", prop.name);
    }
    std::printf("- CUDA managed memory support: %s\n", prop.managedMemory == 1 ? "yes" : "no");
    std::printf("- CUDA unified memory support: %s\n", prop.unifiedAddressing == 1 ? "yes" : "no");
#endif

    test_equivalence();
    test_growing_memory_usage();

    std::printf("All tests passed... Unbelievable!\n");
    return 0;
}
